//model 2 -- 6.1 bank conflict in shared memory
//连续conflictnum数的thread访问同一个array index
//Threads in consecutive conflictnum access the index of an array
//index = i * conflictnum ,  i = 1,2,3,4,5,6…… 
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define DATATYPE int
#define SMEMSIZE 1024
#define REP 128
#define conflictnum 8

__global__ void shared_model_1(double *time,DATATYPE *in1,DATATYPE *in2,DATATYPE *out,int its)
{
	__shared__ DATATYPE smem1[SMEMSIZE];
	__shared__ DATATYPE smem2[SMEMSIZE];
	unsigned int tid=threadIdx.x;
	while(tid<SMEMSIZE)
	{
		smem1[tid]=in1[tid];
		smem2[tid]=in2[tid];
		tid+=blockDim.x;
	}
//	DATATYPE p,q=(threadIdx.x/conflictnum);
//连续conflictnum数的thread访问同一个array index
	DATATYPE p,q=(threadIdx.x/conflictnum*conflictnum);
	double time_tmp=0.0;
	unsigned int start_time=0,stop_time=0;
	unsigned int i,j;
	for (i=0;i<its;i++)
	{
		__syncthreads();
		start_time=clock();
#pragma unroll
		for (j=0;j<REP;j++)
		{
			p=smem1[q];
			q=smem2[p];
		}
		stop_time=clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/REP/its;
	out[blockDim.x*blockIdx.x+threadIdx.x] = p+q;
	time[blockDim.x*blockIdx.x+threadIdx.x] = time_tmp;
}

int main_test(int blocks,int threads,DATATYPE *h_in1,DATATYPE *h_in2)
{
	int its=30;
	//int blocks=1,threads=32;
	DATATYPE *d_in1,*d_in2;
	hipMalloc((void**)&d_in1,sizeof(DATATYPE)*SMEMSIZE);
	hipMalloc((void**)&d_in2,sizeof(DATATYPE)*SMEMSIZE);
	hipMemcpy(d_in1,h_in1,sizeof(DATATYPE)*SMEMSIZE,hipMemcpyHostToDevice);
	hipMemcpy(d_in2,h_in2,sizeof(DATATYPE)*SMEMSIZE,hipMemcpyHostToDevice);
	double *h_time,*d_time;
	DATATYPE *d_out;
	h_time=(double*)malloc(sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_time,sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_out,sizeof(DATATYPE)*blocks*threads);

	shared_model_1<<<blocks,threads>>>(d_time,d_in1,d_in1,d_out,its);
	hipMemcpy(h_time,d_time,sizeof(double)*blocks*threads,hipMemcpyDeviceToHost);
	double avert=0.0,maxt=0.0,mint=99999.9;
	int nn=0;
	for (int i=0;i<blocks;i++)
	{
		for (int j=0;j<threads;j+=32)
		{
			avert+=h_time[i*threads+j];
			nn++;
			if (maxt<h_time[i*threads+j])
			{
				maxt=h_time[i*threads+j];
			}
			if (mint>h_time[i*threads+j])
			{
				mint=h_time[i*threads+j];
			}
		}
	}
	avert/=nn;
	printf("%d\t%d\t\t%f\t%f\t%f\n", blocks,threads,avert,mint,maxt);
	hipFree(d_time);
	hipFree(d_out);
	hipFree(d_in1);
	hipFree(d_in2);
	free(h_time);
	return 0;
}
void init_order(DATATYPE *a,int n)
{
	for (int i=0;i<n;i++)
	{
		a[i]=i;
	}
}

int main()
{
	DATATYPE *h_in1;
	h_in1=(DATATYPE*)malloc(sizeof(DATATYPE)*SMEMSIZE);

	init_order(h_in1,SMEMSIZE);


/*
	for (int i=0;i<SMEMSIZE;i+=32)
	{
		for (int j=0;j<32;j++)
		{
			printf("%d\t",h_in3[i+j]);
		}
		printf("\n");
	}
*/

	printf("blocks\t threads\t aver \t min \t max \t(clocks)\n");

	//main_test(1,32,h_in1,h_in1,1);
	//main_test(1,32,h_in2,h_in2,2);
	//main_test(1,32,h_in3,h_in3,3);
	//main_test(1,512,h_in1,h_in1,1);
	//main_test(1,512,h_in2,h_in2,2);
	//main_test(1,512,h_in3,h_in3,3);



	for (int i=0;i<=1;i+=32)
	{
		int blocks=i;
		if (i==0)
		{
			blocks=1;
		}
		for (int j=0;j<=512;j+=32)
		{
			int threads=j;
			if (j==0)
			{
				threads=1;
			}
			main_test(blocks,threads,h_in1,h_in1);
		}
	}




/*
	for (int i=0;i<=1024;i+=32)
	{
		int blocks=i;
		if (i==0)
		{
			blocks=1;
		}
		for (int j=256;j<=256;j+=32)
		{
			int threads=j;
			if (j==0)
			{
				threads=1;
			}
			main_test(blocks,threads,h_in1,h_in1);
		}
	}
*/


	free(h_in1);

	return 0;
}
