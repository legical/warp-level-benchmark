//Accessing constraints testings
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define DATATYPE int
#define SMEMSIZE 512
#define REP 128

__global__ void shared_model_1(double *time,DATATYPE *in1,DATATYPE *in2,DATATYPE *out,int its)
{
	__shared__ DATATYPE smem1[SMEMSIZE];
	__shared__ DATATYPE smem2[SMEMSIZE];
	unsigned int tid=threadIdx.x;
	while(tid<SMEMSIZE)
	{
		smem1[tid]=in1[tid];
		smem2[tid]=in2[tid];
		tid+=blockDim.x;
	}
	DATATYPE p,q=threadIdx.x;
	double time_tmp=0.0;
	unsigned int start_time=0,stop_time=0;
	unsigned int i,j;
	for (i=0;i<its;i++)
	{
		__syncthreads();
		start_time=clock();
#pragma unroll
		for (j=0;j<REP;j++)
		{
			p=smem1[q];
			q=smem2[p];
		}
		stop_time=clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/REP/its;
	out[blockDim.x*blockIdx.x+threadIdx.x] = p+q;
	time[blockDim.x*blockIdx.x+threadIdx.x] = time_tmp;
}

int main_test(int blocks,int threads,DATATYPE *h_in1,DATATYPE *h_in2,int xxx)
{
	int its=30;
	//int blocks=1,threads=32;
	DATATYPE *d_in1,*d_in2;
	hipMalloc((void**)&d_in1,sizeof(DATATYPE)*SMEMSIZE);
	hipMalloc((void**)&d_in2,sizeof(DATATYPE)*SMEMSIZE);
	hipMemcpy(d_in1,h_in1,sizeof(DATATYPE)*SMEMSIZE,hipMemcpyHostToDevice);
	hipMemcpy(d_in2,h_in2,sizeof(DATATYPE)*SMEMSIZE,hipMemcpyHostToDevice);
	double *h_time,*d_time;
	DATATYPE *d_out;
	h_time=(double*)malloc(sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_time,sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_out,sizeof(DATATYPE)*blocks*threads);

	shared_model_1<<<blocks,threads>>>(d_time,d_in1,d_in1,d_out,its);
	hipMemcpy(h_time,d_time,sizeof(double)*blocks*threads,hipMemcpyDeviceToHost);
	double avert=0.0,maxt=0.0,mint=99999.9;
	int nn=0;
	for (int i=0;i<blocks;i++)
	{
		for (int j=0;j<threads;j+=32)
		{
			avert+=h_time[i*threads+j];
			nn++;
			if (maxt<h_time[i*threads+j])
			{
				maxt=h_time[i*threads+j];
			}
			if (mint>h_time[i*threads+j])
			{
				mint=h_time[i*threads+j];
			}
		}
	}
	avert/=nn;
	printf("%d\t%d\t%d\t\t%f\t%f\t%f\n",xxx, blocks,threads,avert,mint,maxt);
	hipFree(d_time);
	hipFree(d_out);
	hipFree(d_in1);
	hipFree(d_in2);
	free(h_time);
	return 0;
}

//one to one order | case 1
void init_order(DATATYPE *a,int n)
{
	for (int i=0;i<n;i++)
	{
		a[i]=i;
	}
}

//consecutive but misaligned | case 2
void init_disordered_32(DATATYPE *a,int n)
{
	//initial array p (order)
	DATATYPE p[32];
	for (int i=0;i<32;i++)
	{
		p[i]=i;
	}

	for (int i=0;i<n;i+=32)
	{
		for (int j=0;j<32;j++)
		{
			//jj = 0 ~ 32-j-1 (random 0~31)
			int jj=rand()%(32-j);
			//a every 32 index, is random 0~31
			a[i+j]=p[jj];
			//p[jj] has been allocated, so use p[jj+1] cover p[jj]
			for (int k=jj;k<(32-j);k++)
			{
				p[k]=p[k+1];
			}
		}
		for (int j=0;j<32;j++)
		{
			//let p_array initial 0~31
			p[j]=a[i+j];
			//a[i+j] = i + (random 0~31)
			a[i+j]+=i;
		}
	}
}


void init_disordered_512(DATATYPE *a,int n)
{
	const int nn=n/32;
	DATATYPE *q=(DATATYPE*)malloc(sizeof(DATATYPE)*nn);
	DATATYPE *b=(DATATYPE*)malloc(sizeof(DATATYPE)*n);
	init_order(q,nn);
/*
	for (int j=0;j<nn;j++)
	{
		printf("%d\t",q[j]);
	}
	printf("\n");
*/
	for (int i=0;i<n;i+=nn)
	{
		for (int j=0;j<nn;j++)
		{
			//use random q[jj] to fill every 32 elements of b
			//nn-j is decending, so q[jj] is getting smaller
			int jj=rand()%(nn-j);
			b[i+j]=q[jj];
			for (int k=jj;k<(nn-j);k++)
			{
				//The already used q[k] will be overwritten by the highly indexed element
				q[k]=q[k+1];
			}
		}

		for (int j=0;j<nn;j++)
		{
			//q[] repossesses a random 32 elements from b[i+j]
			q[j]=b[i+j];
		}
	}
/*
	for (int i=0;i<n;i+=nn)
	{
		for (int j=0;j<nn;j++)
		{
			printf("%d\t",b[i+j]);
		}
		printf("\n");
	}
*/

	DATATYPE p[32];
	for (int i=0;i<32;i++)
	{
		p[i]=i;
	}
	for (int i=0;i<32;i++)
	{
		for (int j=0;j<nn;j++)
		{
			a[j*32+i]=b[i*nn+j]*32+p[i];
		}
	}
	free(q);
	free(b);
}

int main()
{
	DATATYPE *h_in1,*h_in2,*h_in3;
	h_in1=(DATATYPE*)malloc(sizeof(DATATYPE)*SMEMSIZE);
	h_in2=(DATATYPE*)malloc(sizeof(DATATYPE)*SMEMSIZE);
	h_in3=(DATATYPE*)malloc(sizeof(DATATYPE)*SMEMSIZE);
	init_order(h_in1,SMEMSIZE);
	init_disordered_32(h_in2,SMEMSIZE);
	init_disordered_512(h_in3,SMEMSIZE);

/*
	for (int i=0;i<SMEMSIZE;i+=32)
	{
		for (int j=0;j<32;j++)
		{
			printf("%d\t",h_in3[i+j]);
		}
		printf("\n");
	}
*/

	printf("blocks\t threads\t aver \t min \t max \t(clocks)\n");

	//main_test(1,32,h_in1,h_in1,1);
	//main_test(1,32,h_in2,h_in2,2);
	//main_test(1,32,h_in3,h_in3,3);
	//main_test(1,512,h_in1,h_in1,1);
	//main_test(1,512,h_in2,h_in2,2);
	//main_test(1,512,h_in3,h_in3,3);


/*
	for (int i=0;i<=1;i+=32)
	{
		int blocks=i;
		if (i==0)
		{
			blocks=1;
		}
		for (int j=0;j<=512;j+=32)
		{
			int threads=j;
			if (j==0)
			{
				threads=1;
			}
			main_test(blocks,threads,h_in1,h_in1,1);
			main_test(blocks,threads,h_in2,h_in2,2);
			main_test(blocks,threads,h_in3,h_in3,3);
		}
	}
*/



	for (int i=0;i<=1024;i+=32)
	{
		int blocks=i;
		if (i==0)
		{
			blocks=1;
		}
		for (int j=256;j<=256;j+=32)
		{
			int threads=j;
			if (j==0)
			{
				threads=1;
			}
			main_test(blocks,threads,h_in1,h_in1,1);
			main_test(blocks,threads,h_in2,h_in2,2);
			main_test(blocks,threads,h_in3,h_in3,3);
		}
	}


	free(h_in1);
	free(h_in2);
	free(h_in3);

	return 0;
}
