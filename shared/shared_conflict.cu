//Tesla k20c 
// L1 Cache	16 KB (per SMX)
// L2 Cache	1280 KB
// SMX Count	13
// Memory Size	5 GB
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define DATATYPE int
#define SMEMSIZE 1024
#define REP 128
//bank conflict degree
#define conflictnum 1	//2 4 8 16 32

//input:  int *in1,int *in2,int its
//output: int *out,double *time
__global__ void shared_model_1(double *time,DATATYPE *in1,DATATYPE *in2,DATATYPE *out,int its)
{
	__shared__ DATATYPE smem1[SMEMSIZE];
	__shared__ DATATYPE smem2[SMEMSIZE];

	//threadIdx per block 
	unsigned int tid=threadIdx.x;
	//initial array
	while(tid<SMEMSIZE)
	{
		smem1[tid]=in1[tid];
		smem2[tid]=in2[tid];
		tid+=blockDim.x;
	}
	//q is index of the accessed array in shared memory
	//连续thread的q相差32，每conflictnum重复+1
	//连续thread访问同一个shared memory bank
	DATATYPE p,q=(threadIdx.x%conflictnum)*32+(threadIdx.x/conflictnum);
	double time_tmp=0.0;
	unsigned int start_time=0,stop_time=0;
	unsigned int i,j;
	for (i=0;i<its;i++)
	{
		//块内线程同步
		//确保线程块中的每个线程都执行完 __syncthreads()前面的语句后，才会执行下一条语句
		__syncthreads();
		start_time=clock();
#pragma unroll //没指定次数，对于常数次的循环，循环将完全展开，对于不确定次数的循环，循环将不会被展开
		for (j=0;j<REP;j++)
		{
			p=smem1[q];
			q=smem2[p];
		}
		stop_time=clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/REP/its;
	//calculate the results of p+q to avoid compiler optimizations
	out[blockDim.x*blockIdx.x+threadIdx.x] = p+q;
	time[blockDim.x*blockIdx.x+threadIdx.x] = time_tmp;
}

int main_test(int blocks,int threads,DATATYPE *h_in1,DATATYPE *h_in2)
{
	int its=30;
	//int blocks=1,threads=32;
	//分配array显存
	DATATYPE *d_in1,*d_in2;
	hipMalloc((void**)&d_in1,sizeof(DATATYPE)*SMEMSIZE);
	hipMalloc((void**)&d_in2,sizeof(DATATYPE)*SMEMSIZE);

	//copy数据到GPU上的array
	hipMemcpy(d_in1,h_in1,sizeof(DATATYPE)*SMEMSIZE,hipMemcpyHostToDevice);
	hipMemcpy(d_in2,h_in2,sizeof(DATATYPE)*SMEMSIZE,hipMemcpyHostToDevice);

	//each thread has it's own time
	double *h_time,*d_time;
	DATATYPE *d_out;
	h_time=(double*)malloc(sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_time,sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_out,sizeof(DATATYPE)*blocks*threads);

	//d_time 记录每个thread的执行时间
	shared_model_1<<<blocks,threads>>>(d_time,d_in1,d_in1,d_out,its);	
	hipMemcpy(h_time,d_time,sizeof(double)*blocks*threads,hipMemcpyDeviceToHost);

	//统计threads执行时间的平均值和最值
	double avert=0.0,maxt=0.0,mint=99999.9;
	int nn=0;
	for (int i=0;i<blocks;i++)
	{
		for (int j=0;j<threads;j+=32)
		{
			avert+=h_time[i*threads+j];
			nn++;
			if (maxt<h_time[i*threads+j])
			{
				maxt=h_time[i*threads+j];
			}
			if (mint>h_time[i*threads+j])
			{
				mint=h_time[i*threads+j];
			}
		}
	}
	avert/=nn;

	//打印blocks、threads数，执行时间的平均值和最值
	//printf("blocks number：%d\t threads number：%d\t\t avert Exeu_time：%f\t mint Exeu_time：%f\t maxt Exeu_time：%f\n", blocks,threads,avert,mint,maxt);
	printf("%d\t %d\t\t %f\t %f\t %f\n", blocks,threads,avert,mint,maxt);
	hipFree(d_time);
	hipFree(d_out);
	hipFree(d_in1);
	hipFree(d_in2);
	free(h_time);
	return 0;
}

//初始化数组，a[i]=i
void init_order(DATATYPE *a,int n)
{
	for (int i=0;i<n;i++)
	{
		a[i]=i;
	}
}

int main()
{
	//初始化shared memory大小的数组h_in1
	DATATYPE *h_in1;
	h_in1=(DATATYPE*)malloc(sizeof(DATATYPE)*SMEMSIZE);

	init_order(h_in1,SMEMSIZE);


/*
	for (int i=0;i<SMEMSIZE;i+=32)
	{
		for (int j=0;j<32;j++)
		{
			printf("%d\t",h_in3[i+j]);
		}
		printf("\n");
	}
*/
	printf("conflictnum is： %d\n",conflictnum);
	printf("blocks\t threads\t aver \t\t min \t\t max \t(clocks)\n");

	//main_test(1,32,h_in1,h_in1,1);
	//main_test(1,32,h_in2,h_in2,2);
	//main_test(1,32,h_in3,h_in3,3);
	//main_test(1,512,h_in1,h_in1,1);
	//main_test(1,512,h_in2,h_in2,2);
	//main_test(1,512,h_in3,h_in3,3);



	for (int i=0;i<=1;i+=32)
	{
		int blocks=i;
		if (i==0)
		{
			blocks=1;
		}
		for (int j=0;j<=512;j+=32)
		{
			int threads=j;
			if (j==0)
			{
				threads=1;
			}
			main_test(blocks,threads,h_in1,h_in1);
		}
	}




/*
	for (int i=0;i<=1024;i+=32)
	{
		int blocks=i;
		if (i==0)
		{
			blocks=1;
		}
		for (int j=256;j<=256;j+=32)
		{
			int threads=j;
			if (j==0)
			{
				threads=1;
			}
			main_test(blocks,threads,h_in1,h_in1);
		}
	}
*/


	free(h_in1);

	return 0;
}
