#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define DATATYPE int
#define SMEMSIZE 1024
#define REP 128
//The higher the conflictnum, the greater the broadcasting degree
#define conflictnum 32

__global__ void global_broadcast(double *time,const DATATYPE *in1,const DATATYPE *in2,DATATYPE *out,int its)
{
	unsigned int tid=threadIdx.x;
	//consecutive conflictnum of threads access the same array index
	DATATYPE p,q=(threadIdx.x/conflictnum*conflictnum);
	double time_tmp=0.0;
	unsigned int start_time=0,stop_time=0;
	unsigned int i,j;
	for (i=0;i<its;i++)
	{
		__syncthreads();
		start_time=clock();
#pragma unroll
		for (j=0;j<REP;j++)
		{
			p=in1[q];
			q=in2[p];
		}
		stop_time=clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/REP/its;
	out[blockDim.x*blockIdx.x+threadIdx.x] = p+q;
	time[blockDim.x*blockIdx.x+threadIdx.x] = time_tmp;
}

int main_test(int blocks,int threads,DATATYPE *h_in1,DATATYPE *h_in2)
{
	int its=30;
	//int blocks=1,threads=32;
	DATATYPE *d_in1,*d_in2;
	hipMalloc((void**)&d_in1,sizeof(DATATYPE)*SMEMSIZE);
	hipMalloc((void**)&d_in2,sizeof(DATATYPE)*SMEMSIZE);
	hipMemcpy(d_in1,h_in1,sizeof(DATATYPE)*SMEMSIZE,hipMemcpyHostToDevice);
	hipMemcpy(d_in2,h_in2,sizeof(DATATYPE)*SMEMSIZE,hipMemcpyHostToDevice);
	double *h_time,*d_time;
	DATATYPE *d_out;
	h_time=(double*)malloc(sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_time,sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_out,sizeof(DATATYPE)*blocks*threads);

	global_broadcast<<<blocks,threads>>>(d_time,d_in1,d_in1,d_out,its);
	hipMemcpy(h_time,d_time,sizeof(double)*blocks*threads,hipMemcpyDeviceToHost);
	double avert=0.0,maxt=0.0,mint=99999.9;
	int nn=0;
	for (int i=0;i<blocks;i++)
	{
		for (int j=0;j<threads;j+=32)
		{
			avert+=h_time[i*threads+j];
			nn++;
			if (maxt<h_time[i*threads+j])
			{
				maxt=h_time[i*threads+j];
			}
			if (mint>h_time[i*threads+j])
			{
				mint=h_time[i*threads+j];
			}
		}
	}
	avert/=nn;
	printf("%d\t%d\t\t%f\t%f\t%f\n", blocks,threads,avert,mint,maxt);
	hipFree(d_time);
	hipFree(d_out);
	hipFree(d_in1);
	hipFree(d_in2);
	free(h_time);
	return 0;
}
void init_order(DATATYPE *a,int n)
{
	for (int i=0;i<n;i++)
	{
		a[i]=i;
	}
}

int main()
{
	DATATYPE *h_in1;
	h_in1=(DATATYPE*)malloc(sizeof(DATATYPE)*SMEMSIZE);

	init_order(h_in1,SMEMSIZE);


/*
	for (int i=0;i<SMEMSIZE;i+=32)
	{
		for (int j=0;j<32;j++)
		{
			printf("%d\t",h_in3[i+j]);
		}
		printf("\n");
	}
*/

	printf("blocks\t threads\t aver \t min \t max \t(clocks)\n");

	//main_test(1,32,h_in1,h_in1,1);
	//main_test(1,32,h_in2,h_in2,2);
	//main_test(1,32,h_in3,h_in3,3);
	//main_test(1,512,h_in1,h_in1,1);
	//main_test(1,512,h_in2,h_in2,2);
	//main_test(1,512,h_in3,h_in3,3);



	for (int i=0;i<=1;i+=32)
	{
		int blocks=i;
		if (i==0)
		{
			blocks=1;
		}
		for (int j=0;j<=512;j+=32)
		{
			int threads=j;
			if (j==0)
			{
				threads=1;
			}
			main_test(blocks,threads,h_in1,h_in1);
		}
	}




/*
	for (int i=0;i<=1024;i+=32)
	{
		int blocks=i;
		if (i==0)
		{
			blocks=1;
		}
		for (int j=256;j<=256;j+=32)
		{
			int threads=j;
			if (j==0)
			{
				threads=1;
			}
			main_test(blocks,threads,h_in1,h_in1);
		}
	}
*/


	free(h_in1);

	return 0;
}
